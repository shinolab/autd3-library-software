#include "hip/hip_runtime.h"
// File: cuda_backend.cpp
// Project: cuda
// Created Date: 04/09/2021
// Author: Shun Suzuki
// -----
// Last Modified: 06/09/2021
// Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
// -----
// Copyright (c) 2021 Hapis Lab. All rights reserved.
//

#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/reduce.h>

#include "./kernel.h"
#include "autd3/gain/cuda_backend.hpp"

namespace {
hipblasOperation_t convert(autd::gain::holo::TRANSPOSE trans) {
  switch (trans) {
    case autd::gain::holo::TRANSPOSE::NO_TRANS:
      return hipblasOperation_t::HIPBLAS_OP_N;
    case autd::gain::holo::TRANSPOSE::CONJ_TRANS:
      return hipblasOperation_t::HIPBLAS_OP_C;
    case autd::gain::holo::TRANSPOSE::TRANS:
      return hipblasOperation_t::HIPBLAS_OP_T;
  }
  return hipblasOperation_t::HIPBLAS_OP_N;
}
}  // namespace

namespace autd {
namespace gain {
namespace holo {

template <typename T>
struct CuMatrix final : public Matrix<T> {
  explicit CuMatrix(const Eigen::Index row, const Eigen::Index col) : Matrix<T>(row, col), _row(row), _col(col), _d_vec(_row * _col) {}
  ~CuMatrix() override {}
  CuMatrix(const CuMatrix& obj) = delete;
  CuMatrix& operator=(const CuMatrix& obj) = delete;
  CuMatrix(const CuMatrix&& v) = delete;
  CuMatrix& operator=(CuMatrix&& obj) = delete;

  [[nodiscard]] T at(size_t row, size_t col) const override {
    T r;
    hipMemcpy(&r, (const void*)(&_d_vec[row + col * _row]).get(), sizeof(T), hipMemcpyDeviceToHost);
    return r;
  };
  const T* ptr() const override { return _d_vec.data().get(); }
  T* ptr() override { return _d_vec.data().get(); }
  [[nodiscard]] double max_element() const override;
  void set(const Eigen::Index row, const Eigen::Index col, T v) override {
    hipMemcpy((void*)(&_d_vec[row + col * _row]).get(), &v, sizeof(T), hipMemcpyHostToDevice);
  }
  void get_col(const Eigen::Index i, std::shared_ptr<Matrix<T>> dst) override {
    hipMemcpy(dst->ptr(), ptr() + i * _row, _row * sizeof(T), hipMemcpyDeviceToDevice);
  }
  void fill(T v) override { thrust::fill(_d_vec.begin(), _d_vec.end(), v); }
  void get_diagonal(std::shared_ptr<Matrix<T>> v) override { cu_get_diagonal(ptr(), v->ptr(), (uint32_t)(std::min)(data.rows(), data.cols())); }
  void set_diagonal(std::shared_ptr<Matrix<T>> v) override { cu_set_diagonal(v->ptr(), ptr(), (uint32_t)(std::min)(data.rows(), data.cols())); }
  void copy_from(const std::vector<T>& v) override { hipMemcpy(_d_vec.data().get(), v.data(), v.size() * sizeof(T), hipMemcpyHostToDevice); }
  void copy_from(const T* v) override { hipMemcpy(_d_vec.data().get(), v, _d_vec.size() * sizeof(T), hipMemcpyHostToDevice); }
  void copy_to_host() override { hipMemcpy(data.data(), _d_vec.data().get(), _row * _col * sizeof(T), hipMemcpyDeviceToHost); }

 private:
  Eigen::Index _row;
  Eigen::Index _col;
  thrust::device_vector<T> _d_vec;
};

template <>
double CuMatrix<double>::max_element() const {
  return *thrust::max_element(_d_vec.begin(), _d_vec.end());
}

template <>
double CuMatrix<complex>::max_element() const {
  throw std::runtime_error("not impletemted max_element for complex");
}

CUDABackend::CUDABackend() {
  hipblasCreate(&_handle);
  hipsolverDnCreate(&_handle_s);
}
CUDABackend::~CUDABackend() {
  hipblasDestroy(_handle);
  hipsolverDnDestroy(_handle_s);
}

template <typename T, typename C>
static std::shared_ptr<T> allocate_cu_matrix_impl(const std::string& name, const int64_t row, const int64_t col,
                                                  std::unordered_map<std::string, std::shared_ptr<T>>& cache) {
  const auto it = cache.find(name);
  if (it != cache.end()) {
    if (it->second->data.rows() == row && it->second->data.cols() == col) return it->second;
    cache.erase(name);
  }
  auto v = std::make_shared<C>(row, col);
  cache.emplace(name, v);
  return v;
}

std::shared_ptr<MatrixX> CUDABackend::allocate_matrix(const std::string& name, const size_t row, const size_t col) {
  return allocate_cu_matrix_impl<MatrixX, CuMatrix<double>>(name, row, col, _cache_mat);
}

std::shared_ptr<MatrixXc> CUDABackend::allocate_matrix_c(const std::string& name, const size_t row, const size_t col) {
  return allocate_cu_matrix_impl<MatrixXc, CuMatrix<complex>>(name, row, col, _cache_mat_c);
}

BackendPtr CUDABackend::create() { return std::make_shared<CUDABackend>(); }

void CUDABackend::make_complex(const std::shared_ptr<MatrixX> r, const std::shared_ptr<MatrixX> i, const std::shared_ptr<MatrixXc> c) {
  cu_make_complex(r->ptr(), i->ptr(), (uint32_t)c->data.rows(), (uint32_t)c->data.cols(), (hipDoubleComplex*)c->ptr());
}
void CUDABackend::exp(const std::shared_ptr<MatrixXc> a) { cu_exp((uint32_t)a->data.rows(), (uint32_t)a->data.cols(), (hipDoubleComplex*)a->ptr()); }
void CUDABackend::scale(const std::shared_ptr<MatrixXc> a, const complex s) {
  hipblasZscal(_handle, static_cast<int>(a->data.size()), (const hipDoubleComplex*)&s, (hipDoubleComplex*)a->ptr(), 1);
}
void CUDABackend::hadamard_product(const std::shared_ptr<MatrixXc> a, const std::shared_ptr<MatrixXc> b, const std::shared_ptr<MatrixXc> c) {
  cu_hadamard_product((const hipDoubleComplex*)a->ptr(), (const hipDoubleComplex*)b->ptr(), (uint32_t)c->data.rows(), (uint32_t)c->data.cols(),
                      (hipDoubleComplex*)c->ptr());
}
void CUDABackend::real(const std::shared_ptr<MatrixXc> a, const std::shared_ptr<MatrixX> b) {
  cu_real((const hipDoubleComplex*)a->ptr(), (uint32_t)a->data.rows(), (uint32_t)a->data.cols(), b->ptr());
}
void CUDABackend::arg(const std::shared_ptr<MatrixXc> a, const std::shared_ptr<MatrixXc> c) {
  cu_arg((const hipDoubleComplex*)a->ptr(), (uint32_t)a->data.rows(), (uint32_t)a->data.cols(), (hipDoubleComplex*)c->ptr());
}
void CUDABackend::pseudo_inverse_svd(const std::shared_ptr<MatrixXc> matrix, const double alpha, const std::shared_ptr<MatrixXc> result) {
  const auto nc = matrix->data.cols();
  const auto nr = matrix->data.rows();

  const auto lda = static_cast<int>(nr);
  const auto ldu = static_cast<int>(nr);
  const auto ldv = static_cast<int>(nc);

  const auto s_size = std::min(nr, nc);
  double* d_s = nullptr;
  hipMalloc((void**)&d_s, sizeof(double) * s_size);
  const auto a = this->allocate_matrix_c("_pis_a", matrix->data.rows(), matrix->data.cols());
  const auto u = this->allocate_matrix_c("_pis_u", nr, nr);
  const auto v = this->allocate_matrix_c("_pis_v", nc, nc);
  hipMemcpy(a->ptr(), matrix->ptr(), matrix->data.rows() * matrix->data.cols() * sizeof(complex), hipMemcpyDeviceToDevice);

  size_t workspace_in_bytes_on_device;
  size_t workspace_in_bytes_on_host;

  cusolverDnXgesvdp_bufferSize(_handle_s, NULL, hipsolverEigMode_t::HIPSOLVER_EIG_MODE_VECTOR, 0, static_cast<int>(nr), static_cast<int>(nc),
                               hipDataType::HIP_C_64F, a->ptr(), lda, hipDataType::HIP_R_64F, d_s, hipDataType::HIP_C_64F, u->ptr(), ldu,
                               hipDataType::HIP_C_64F, v->ptr(), ldv, hipDataType::HIP_C_64F, &workspace_in_bytes_on_device,
                               &workspace_in_bytes_on_host);
  void* workspace_buffer_on_device = nullptr;
  void* workspace_buffer_on_host = nullptr;
  hipMalloc((void**)&workspace_buffer_on_device, workspace_in_bytes_on_device);
  if (workspace_in_bytes_on_host > 0) workspace_buffer_on_host = (void*)malloc(workspace_in_bytes_on_host);

  int* info;
  hipMalloc((void**)&info, sizeof(int));
  double h_err_sigma;
  cusolverDnXgesvdp(_handle_s, NULL, hipsolverEigMode_t::HIPSOLVER_EIG_MODE_VECTOR, 0, static_cast<int>(nr), static_cast<int>(nc),
                    hipDataType::HIP_C_64F, a->ptr(), lda, hipDataType::HIP_R_64F, d_s, hipDataType::HIP_C_64F, u->ptr(), ldu,
                    hipDataType::HIP_C_64F, v->ptr(), ldv, hipDataType::HIP_C_64F, workspace_buffer_on_device, workspace_in_bytes_on_device,
                    workspace_buffer_on_host, workspace_in_bytes_on_host, info, &h_err_sigma);

  const auto singular_inv = this->allocate_matrix_c("_pis_si", nc, nr);
  calc_singular_inv(d_s, (uint32_t)s_size, alpha, (hipDoubleComplex*)singular_inv->ptr());

  const auto tmp = this->allocate_matrix_c("_pis_tmp", nc, nr);
  CUDABackend::matrix_mul(TRANSPOSE::NO_TRANS, TRANSPOSE::CONJ_TRANS, ONE, singular_inv, u, ZERO, tmp);
  CUDABackend::matrix_mul(TRANSPOSE::NO_TRANS, TRANSPOSE::NO_TRANS, ONE, v, tmp, ZERO, result);
  hipFree(d_s);
  hipFree(info);
  hipFree(workspace_buffer_on_device);
  free(workspace_buffer_on_host);
}
void CUDABackend::max_eigen_vector(const std::shared_ptr<MatrixXc> matrix, const std::shared_ptr<MatrixXc> ev) {
  const auto size = matrix->data.cols();

  double* d_w = nullptr;
  hipMalloc((void**)&d_w, sizeof(double) * size);

  size_t workspace_in_bytes_on_device;
  size_t workspace_in_bytes_on_host;
  cusolverDnXsyevd_bufferSize(_handle_s, NULL, hipsolverEigMode_t::HIPSOLVER_EIG_MODE_VECTOR, hipblasFillMode_t::HIPBLAS_FILL_MODE_UPPER, size,
                              hipDataType::HIP_C_64F, matrix->ptr(), size, hipDataType::HIP_R_64F, d_w, hipDataType::HIP_C_64F,
                              &workspace_in_bytes_on_device, &workspace_in_bytes_on_host);

  void* workspace_buffer_on_device = nullptr;
  void* workspace_buffer_on_host = nullptr;
  hipMalloc((void**)&workspace_buffer_on_device, workspace_in_bytes_on_device);
  if (workspace_in_bytes_on_host > 0) workspace_buffer_on_host = (void*)malloc(workspace_in_bytes_on_host);

  int* info;
  hipMalloc((void**)&info, sizeof(int));
  cusolverDnXsyevd(_handle_s, NULL, hipsolverEigMode_t::HIPSOLVER_EIG_MODE_VECTOR, hipblasFillMode_t::HIPBLAS_FILL_MODE_UPPER, size,
                   hipDataType::HIP_C_64F, matrix->ptr(), size, hipDataType::HIP_R_64F, d_w, hipDataType::HIP_C_64F, workspace_buffer_on_device,
                   workspace_in_bytes_on_device, workspace_buffer_on_host, workspace_in_bytes_on_host, info);
  hipFree(d_w);
  hipFree(info);
  hipFree(workspace_buffer_on_device);
  free(workspace_buffer_on_host);

  hipMemcpy(ev->ptr(), matrix->ptr() + size * (size - 1), size * sizeof(complex), hipMemcpyDeviceToDevice);
}

void CUDABackend::matrix_add(const double alpha, const std::shared_ptr<MatrixX> a, const std::shared_ptr<MatrixX> b) {
  hipblasDaxpy(_handle, static_cast<int>(a->data.size()), &alpha, a->ptr(), 1, b->ptr(), 1);
}
void CUDABackend::matrix_mul(const TRANSPOSE trans_a, const TRANSPOSE trans_b, const complex alpha, const std::shared_ptr<MatrixXc> a,
                             const std::shared_ptr<MatrixXc> b, const complex beta, const std::shared_ptr<MatrixXc> c) {
  const auto lda = static_cast<int>(a->data.rows());
  const auto ldb = static_cast<int>(b->data.rows());
  const auto ldc = trans_a == TRANSPOSE::NO_TRANS ? static_cast<int>(a->data.rows()) : static_cast<int>(a->data.cols());
  const auto n = trans_b == TRANSPOSE::NO_TRANS ? static_cast<int>(b->data.cols()) : static_cast<int>(b->data.rows());
  const auto k = trans_a == TRANSPOSE::NO_TRANS ? static_cast<int>(a->data.cols()) : static_cast<int>(a->data.rows());
  cublasZgemm3m(_handle, convert(trans_a), convert(trans_b), ldc, n, k, (const hipDoubleComplex*)&alpha, (const hipDoubleComplex*)a->ptr(), lda,
                (const hipDoubleComplex*)b->ptr(), ldb, (const hipDoubleComplex*)&beta, (hipDoubleComplex*)c->ptr(), ldc);
}
void CUDABackend::matrix_mul(const TRANSPOSE trans_a, const TRANSPOSE trans_b, const double alpha, const std::shared_ptr<MatrixX> a,
                             const std::shared_ptr<MatrixX> b, const double beta, const std::shared_ptr<MatrixX> c) {
  const auto lda = static_cast<int>(a->data.rows());
  const auto ldb = static_cast<int>(b->data.rows());
  const auto ldc = trans_a == TRANSPOSE::NO_TRANS ? static_cast<int>(a->data.rows()) : static_cast<int>(a->data.cols());
  const auto n = trans_b == TRANSPOSE::NO_TRANS ? static_cast<int>(b->data.cols()) : static_cast<int>(b->data.rows());
  const auto k = trans_a == TRANSPOSE::NO_TRANS ? static_cast<int>(a->data.cols()) : static_cast<int>(a->data.rows());
  hipblasDgemm(_handle, convert(trans_a), convert(trans_b), ldc, n, k, &alpha, a->ptr(), lda, b->ptr(), ldb, &beta, c->ptr(), ldc);
}

void CUDABackend::solve_g(const std::shared_ptr<MatrixX> a, const std::shared_ptr<MatrixX> b, const std::shared_ptr<MatrixX> c) {
  const auto n = static_cast<int>(a->data.cols());
  const auto lda = static_cast<int>(a->data.rows());
  const auto ldb = static_cast<int>(b->data.size());
  mat_cpy(b, c);

  size_t workspace_in_bytes_on_device;
  size_t workspace_in_bytes_on_host;
  hipsolverDnXpotrf_bufferSize(_handle_s, NULL, hipblasFillMode_t::HIPBLAS_FILL_MODE_UPPER, n, hipDataType::HIP_R_64F, a->ptr(), lda,
                              hipDataType::HIP_R_64F, &workspace_in_bytes_on_device, &workspace_in_bytes_on_host);

  void* workspace_buffer_on_device = nullptr;
  void* workspace_buffer_on_host = nullptr;
  hipMalloc((void**)&workspace_buffer_on_device, workspace_in_bytes_on_device);
  if (workspace_in_bytes_on_host > 0) workspace_buffer_on_host = (void*)malloc(workspace_in_bytes_on_host);

  int* info;
  hipMalloc((void**)&info, sizeof(int));
  hipsolverDnXpotrf(_handle_s, NULL, hipblasFillMode_t::HIPBLAS_FILL_MODE_UPPER, n, hipDataType::HIP_R_64F, a->ptr(), lda, hipDataType::HIP_R_64F,
                   workspace_buffer_on_device, workspace_in_bytes_on_device, workspace_buffer_on_host, workspace_in_bytes_on_host, info);

  hipsolverDnXpotrs(_handle_s, NULL, hipblasFillMode_t::HIPBLAS_FILL_MODE_UPPER, n, 1, hipDataType::HIP_R_64F, a->ptr(), lda, hipDataType::HIP_R_64F,
                   c->ptr(), ldb, info);

  hipFree(info);
  hipFree(workspace_buffer_on_device);
  free(workspace_buffer_on_host);
}
void CUDABackend::solve_ch(const std::shared_ptr<MatrixXc> a, const std::shared_ptr<MatrixXc> b) {
  const auto n = static_cast<int>(a->data.cols());
  const auto lda = static_cast<int>(a->data.rows());
  const auto ldb = static_cast<int>(b->data.size());

  size_t workspace_in_bytes_on_device;
  size_t workspace_in_bytes_on_host;
  hipsolverDnXpotrf_bufferSize(_handle_s, NULL, hipblasFillMode_t::HIPBLAS_FILL_MODE_UPPER, n, hipDataType::HIP_C_64F, a->ptr(), lda,
                              hipDataType::HIP_C_64F, &workspace_in_bytes_on_device, &workspace_in_bytes_on_host);

  void* workspace_buffer_on_device = nullptr;
  void* workspace_buffer_on_host = nullptr;
  hipMalloc((void**)&workspace_buffer_on_device, workspace_in_bytes_on_device);
  if (workspace_in_bytes_on_host > 0) workspace_buffer_on_host = (void*)malloc(workspace_in_bytes_on_host);

  int* info;
  hipMalloc((void**)&info, sizeof(int));
  hipsolverDnXpotrf(_handle_s, NULL, hipblasFillMode_t::HIPBLAS_FILL_MODE_UPPER, n, hipDataType::HIP_C_64F, a->ptr(), lda, hipDataType::HIP_C_64F,
                   workspace_buffer_on_device, workspace_in_bytes_on_device, workspace_buffer_on_host, workspace_in_bytes_on_host, info);

  hipsolverDnXpotrs(_handle_s, NULL, hipblasFillMode_t::HIPBLAS_FILL_MODE_UPPER, n, 1, hipDataType::HIP_C_64F, a->ptr(), lda, hipDataType::HIP_C_64F,
                   b->ptr(), ldb, info);

  hipFree(info);
  hipFree(workspace_buffer_on_device);
  free(workspace_buffer_on_host);
}
double CUDABackend::dot(const std::shared_ptr<MatrixX> a, const std::shared_ptr<MatrixX> b) {
  double d;
  hipblasDdot(_handle, static_cast<int>(a->data.size()), a->ptr(), 1, b->ptr(), 1, &d);
  return d;
}
complex CUDABackend::dot(const std::shared_ptr<MatrixXc> a, const std::shared_ptr<MatrixXc> b) {
  complex d;
  hipblasZdotc(_handle, static_cast<int>(a->data.size()), (const hipDoubleComplex*)a->ptr(), 1, (const hipDoubleComplex*)b->ptr(), 1,
                 (hipDoubleComplex*)&d);
  return d;
}
double CUDABackend::max_coefficient(const std::shared_ptr<MatrixXc> v) {
  int idx;
  hipblasIzamax(_handle, static_cast<int>(v->data.size()), (const hipDoubleComplex*)v->ptr(), 1, &idx);
  return std::abs(v->at(idx - 1, 0));  // 1-based indexing
}
double CUDABackend::max_coefficient(const std::shared_ptr<MatrixX> v) { return v->max_element(); }
std::shared_ptr<MatrixXc> CUDABackend::concat_row(const std::shared_ptr<MatrixXc> a, const std::shared_ptr<MatrixXc> b) {
  auto r = allocate_matrix_c("_cc_row", a->data.rows() + b->data.rows(), a->data.cols());
  for (int64_t i = 0; i < a->data.cols(); i++) {
    hipMemcpy(r->ptr() + i * (a->data.rows() + b->data.rows()), a->ptr() + i * a->data.rows(), a->data.rows() * sizeof(complex),
               hipMemcpyDeviceToDevice);
    hipMemcpy(r->ptr() + i * (a->data.rows() + b->data.rows()) + a->data.rows(), b->ptr() + i * b->data.rows(), b->data.rows() * sizeof(complex),
               hipMemcpyDeviceToDevice);
  }
  return r;
}
std::shared_ptr<MatrixXc> CUDABackend::concat_col(const std::shared_ptr<MatrixXc> a, const std::shared_ptr<MatrixXc> b) {
  auto r = allocate_matrix_c("_cc_col", a->data.rows(), a->data.cols() + b->data.cols());
  hipMemcpy(r->ptr(), a->ptr(), a->data.rows() * a->data.cols() * sizeof(complex), hipMemcpyDeviceToDevice);
  hipMemcpy(r->ptr() + a->data.rows() * a->data.cols(), b->ptr(), b->data.rows() * b->data.cols() * sizeof(complex), hipMemcpyDeviceToDevice);
  return r;
}
void CUDABackend::mat_cpy(const std::shared_ptr<MatrixX> a, const std::shared_ptr<MatrixX> b) {
  hipblasDcopy(_handle, static_cast<int>(a->data.rows()) * static_cast<int>(a->data.cols()), a->ptr(), 1, b->ptr(), 1);
}
void CUDABackend::mat_cpy(const std::shared_ptr<MatrixXc> a, const std::shared_ptr<MatrixXc> b) {
  hipblasZcopy(_handle, static_cast<int>(a->data.rows()) * static_cast<int>(a->data.cols()), (const hipDoubleComplex*)a->ptr(), 1,
                 (hipDoubleComplex*)b->ptr(), 1);
}

void CUDABackend::set_from_complex_drive(std::vector<core::DataArray>& data, const std::shared_ptr<MatrixXc> drive, const bool normalize,
                                         const double max_coefficient) {
  uint16_t* d_data = nullptr;
  hipMalloc((void**)&d_data, data.size() * core::NUM_TRANS_IN_UNIT * sizeof(uint16_t));

  cu_set_from_complex_drive((const hipDoubleComplex*)drive->ptr(), (uint32_t)(data.size() * core::NUM_TRANS_IN_UNIT), normalize, max_coefficient,
                            d_data);
  for (size_t i = 0; i < data.size(); i++)
    hipMemcpy(data[i].data(), d_data + i * core::NUM_TRANS_IN_UNIT, core::NUM_TRANS_IN_UNIT * sizeof(uint16_t), hipMemcpyDeviceToHost);
}

std::shared_ptr<MatrixXc> CUDABackend::transfer_matrix(const double* foci, size_t foci_num, const std::vector<const double*>& positions,
                                                       const std::vector<const double*>& directions, double wavelength, double attenuation) {
  const auto m = static_cast<Eigen::Index>(foci_num);
  const auto n = static_cast<Eigen::Index>(positions.size() * core::NUM_TRANS_IN_UNIT);

  auto g = allocate_matrix_c("g", m, n);

  auto d_foci = allocate_matrix("_foci", 3, m);
  auto d_pos = allocate_matrix("_pos", 3, n);
  auto d_dir = allocate_matrix("_dir", 3, directions.size());
  hipMemcpy(d_foci->ptr(), foci, m * 3 * sizeof(double), hipMemcpyHostToDevice);
  for (size_t i = 0; i < positions.size(); i++)
    hipMemcpy(d_pos->ptr() + core::NUM_TRANS_IN_UNIT * 3 * i, positions[i], core::NUM_TRANS_IN_UNIT * 3 * sizeof(double), hipMemcpyHostToDevice);
  for (size_t i = 0; i < directions.size(); i++) hipMemcpy(d_dir->ptr() + 3 * i, directions[i], 3 * sizeof(double), hipMemcpyHostToDevice);

  cu_transfer_matrix((const double3*)d_foci->ptr(), (uint32_t)m, (const double3*)d_pos->ptr(), (const double3*)d_dir->ptr(), (uint32_t)n,
                     2.0 * M_PI / wavelength, attenuation, (hipDoubleComplex*)g->ptr());

  return g;
}

void CUDABackend::set_bcd_result(const std::shared_ptr<MatrixXc> mat, const std::shared_ptr<MatrixXc> vec, const size_t idx) {
  const uint32_t m = (uint32_t)vec->data.size();
  cu_set_bcd_result((const hipDoubleComplex*)vec->ptr(), m, (uint32_t)idx, (hipDoubleComplex*)mat->ptr());
}

std::shared_ptr<MatrixXc> CUDABackend::back_prop(const std::shared_ptr<MatrixXc> transfer, const std::shared_ptr<MatrixXc> amps) {
  const auto m = transfer->data.rows();
  const auto n = transfer->data.cols();

  auto denominator = allocate_matrix("denomi", m, 1);
  auto buffer = allocate_matrix("_bp_buf", m, 16);
  cu_col_sum_abs((const hipDoubleComplex*)transfer->ptr(), (uint32_t)m, (uint32_t)n, denominator->ptr(), buffer->ptr());

  auto b = allocate_matrix_c("b", n, m);
  cu_make_back_prop((const hipDoubleComplex*)amps->ptr(), denominator->ptr(), (const hipDoubleComplex*)transfer->ptr(), (uint32_t)m, (uint32_t)n,
                    (hipDoubleComplex*)b->ptr());
  return b;
}

std::shared_ptr<MatrixXc> CUDABackend::sigma_regularization(const std::shared_ptr<MatrixXc> transfer, const std::shared_ptr<MatrixXc> amps,
                                                            const double gamma) {
  const auto m = transfer->data.rows();
  const auto n = transfer->data.cols();

  auto tmp = allocate_matrix_c("_sr_tmp", n, 1);
  auto buffer = allocate_matrix("_sr_buffer", 16, n);
  cu_make_sigma_diagonal((const hipDoubleComplex*)transfer->ptr(), (uint32_t)m, (uint32_t)n, (const hipDoubleComplex*)amps->ptr(), gamma,
                         (hipDoubleComplex*)tmp->ptr(), buffer->ptr());

  auto sigma = allocate_matrix_c("sigma", n, n);
  sigma->fill(ZERO);
  sigma->set_diagonal(tmp);

  return sigma;
}

void CUDABackend::col_sum_imag(const std::shared_ptr<MatrixXc> mat, const std::shared_ptr<MatrixX> dst) {
  const auto m = mat->data.rows();
  const auto n = mat->data.cols();

  auto buffer = allocate_matrix("_csi_buf", m, 16);
  cu_col_sum_imag((const hipDoubleComplex*)mat->ptr(), (uint32_t)m, (uint32_t)n, dst->ptr(), buffer->ptr());
}

}  // namespace holo
}  // namespace gain
}  // namespace autd
