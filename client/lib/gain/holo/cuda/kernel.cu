#include "hip/hip_runtime.h"
/*
 * File: kernel.cu
 * Project: cuda
 * Created Date: 06/09/2021
 * Author: Shun Suzuki
 * -----
 * Last Modified: 22/09/2021
 * Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
 * -----
 * Copyright (c) 2021 Hapis Lab. All rights reserved.
 *
 */

#include <hip/hip_runtime_api.h>

#include <complex>

#include "./kernel.h"
#include "autd3/core/hardware_defined.hpp"

namespace autd {
namespace gain {
namespace holo {

template <typename T>
__global__ void get_diagonal_kernel(const T* src, T* dst, uint32_t size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) return;
  dst[i] = src[i + size * i];
}

template <>
void cu_get_diagonal(const double* src, double* dst, uint32_t size) {
  dim3 block(BLOCK_SIZE, 1, 1);
  dim3 grid((size - 1) / BLOCK_SIZE + 1, 1, 1);
  get_diagonal_kernel<<<grid, block>>>(src, dst, size);
}

template <>
void cu_get_diagonal(const std::complex<double>* src, std::complex<double>* dst, uint32_t size) {
  dim3 block(BLOCK_SIZE, 1, 1);
  dim3 grid((size - 1) / BLOCK_SIZE + 1, 1, 1);
  get_diagonal_kernel<<<grid, block>>>(src, dst, size);
}

template <typename T>
__global__ void set_diagonal_kernel(const T* src, T* dst, uint32_t size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) return;
  dst[i + size * i] = src[i];
}

template <>
void cu_set_diagonal(const double* src, double* dst, uint32_t size) {
  dim3 block(BLOCK_SIZE, 1, 1);
  dim3 grid((size - 1) / BLOCK_SIZE + 1, 1, 1);
  set_diagonal_kernel<<<grid, block>>>(src, dst, size);
}

template <>
void cu_set_diagonal(const std::complex<double>* src, std::complex<double>* dst, uint32_t size) {
  dim3 block(BLOCK_SIZE, 1, 1);
  dim3 grid((size - 1) / BLOCK_SIZE + 1, 1, 1);
  set_diagonal_kernel<<<grid, block>>>(src, dst, size);
}

__global__ void make_complex_kernel(const double* r, const double* i, const uint32_t row, const uint32_t col, hipDoubleComplex* c) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  c[idx] = make_hipDoubleComplex(r[idx], i[idx]);
}

void cu_make_complex(const double* r, const double* i, const uint32_t row, const uint32_t col, hipDoubleComplex* c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  make_complex_kernel<<<grid, block>>>(r, i, row, col, c);
}

__device__ hipDoubleComplex expc(hipDoubleComplex x) {
  double s = exp(x.x);
  double r = cos(x.y);
  double i = sin(x.y);
  return make_hipDoubleComplex(s * r, s * i);
}

__global__ void exp_kernel(const uint32_t row, const uint32_t col, double* c) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  c[idx] = exp(c[idx]);
}
__global__ void exp_kernel(const uint32_t row, const uint32_t col, hipDoubleComplex* c) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  c[idx] = expc(c[idx]);
}

void cu_exp(const uint32_t row, const uint32_t col, double* c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  exp_kernel<<<grid, block>>>(row, col, c);
}
void cu_exp(const uint32_t row, const uint32_t col, hipDoubleComplex* c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  exp_kernel<<<grid, block>>>(row, col, c);
}

__device__ hipDoubleComplex conj(hipDoubleComplex a) { return make_hipDoubleComplex(a.x, -a.y); }
__device__ double absc2(hipDoubleComplex x) { return x.x * x.x + x.y * x.y; }
__device__ double absc(hipDoubleComplex x) { return sqrt(absc2(x)); }

__global__ void reciprocal_kernel(const uint32_t row, const uint32_t col, const double* src, double* dst) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  dst[idx] = 1.0 / src[idx];
}
__global__ void reciprocal_kernel(const uint32_t row, const uint32_t col, const hipDoubleComplex* src, hipDoubleComplex* dst) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  double d = absc2(src[idx]);
  dst[idx] = make_hipDoubleComplex(src[idx].x / d, -src[idx].y / d);
}

void cu_reciprocal(const uint32_t row, const uint32_t col, const double* src, double* dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  reciprocal_kernel<<<grid, block>>>(row, col, src, dst);
}
void cu_reciprocal(const uint32_t row, const uint32_t col, const hipDoubleComplex* src, hipDoubleComplex* dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  reciprocal_kernel<<<grid, block>>>(row, col, src, dst);
}

__global__ void abs_kernel(const uint32_t row, const uint32_t col, const double* src, double* dst) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  dst[idx] = abs(src[idx]);
}
__global__ void abs_kernel(const uint32_t row, const uint32_t col, const hipDoubleComplex* src, hipDoubleComplex* dst) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  dst[idx] = make_hipDoubleComplex(absc(src[idx]), 0.0);
}

void cu_abs(const uint32_t row, const uint32_t col, const double* src, double* dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(row, col, src, dst);
}
void cu_abs(const uint32_t row, const uint32_t col, const hipDoubleComplex* src, hipDoubleComplex* dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(row, col, src, dst);
}

__device__ hipDoubleComplex mulc(hipDoubleComplex a, hipDoubleComplex b) { return make_hipDoubleComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x); }

__global__ void hadamard_product_kernel(const double* a, const double* b, const uint32_t row, const uint32_t col, double* c) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  c[idx] = a[idx] * b[idx];
}
__global__ void hadamard_product_kernel(const hipDoubleComplex* a, const hipDoubleComplex* b, const uint32_t row, const uint32_t col,
                                        hipDoubleComplex* c) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  c[idx] = mulc(a[idx], b[idx]);
}

void cu_hadamard_product(const double* a, const double* b, uint32_t row, uint32_t col, double* c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  hadamard_product_kernel<<<grid, block>>>(a, b, row, col, c);
}
void cu_hadamard_product(const hipDoubleComplex* a, const hipDoubleComplex* b, uint32_t row, uint32_t col, hipDoubleComplex* c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  hadamard_product_kernel<<<grid, block>>>(a, b, row, col, c);
}

__global__ void real_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, double* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  b[idx] = a[idx].x;
}

void cu_real(const hipDoubleComplex* a, uint32_t row, uint32_t col, double* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  real_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void arg_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  double s = absc(a[idx]);
  b[idx] = make_hipDoubleComplex(a[idx].x / s, a[idx].y / s);
}

void cu_arg(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  arg_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void calc_singular_inv_kernel(double* d_s, uint32_t s_size, double alpha, hipDoubleComplex* p_singular_inv) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= s_size || yi >= s_size) return;

  if (xi != yi)
    p_singular_inv[xi + yi * s_size] = make_hipDoubleComplex(0.0, 0.0);
  else
    p_singular_inv[xi + yi * s_size] = make_hipDoubleComplex(d_s[xi] / (d_s[xi] * d_s[xi] + alpha), 0.0);
}

__global__ void calc_singular_inv_kernel(double* d_s, uint32_t s_size, double alpha, double* p_singular_inv) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= s_size || yi >= s_size) return;

  if (xi != yi)
    p_singular_inv[xi + yi * s_size] = 0.0;
  else
    p_singular_inv[xi + yi * s_size] = d_s[xi] / (d_s[xi] * d_s[xi] + alpha);
}

void calc_singular_inv(double* d_s, uint32_t s_size, double alpha, hipDoubleComplex* p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((s_size - 1) / BLOCK_SIZE + 1, (s_size - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, s_size, alpha, p_singular_inv);
}

void calc_singular_inv(double* d_s, uint32_t s_size, double alpha, double* p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((s_size - 1) / BLOCK_SIZE + 1, (s_size - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, s_size, alpha, p_singular_inv);
}

template <typename T>
__device__ T clamp(T v, T min, T max) {
  return v < min ? min : v > max ? max : v;
}

__device__ uint8_t to_duty(const double amp) {
  const auto d = asin(clamp(amp, 0.0, 1.0)) / M_PI;
  return (uint8_t)(510.0 * d);
}

__device__ uint8_t to_phase(const double phase) noexcept {
  const uint8_t d_phase = (uint8_t)((int)(round((phase / (2.0 * M_PI) + 0.5) * 256.0)) & 0xFF);
  return core::PHASE_INVERTED ? d_phase : 0xFF - d_phase;
}

__global__ void set_from_complex_drive_kernel(const hipDoubleComplex* drive, uint32_t size, bool normalize, double max_coefficient, uint16_t* d_data) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  if (xi >= size) return;

  const auto f_amp = normalize ? 1.0 : absc(drive[xi]) / max_coefficient;
  const uint16_t phase = (uint16_t)to_phase(atan2(drive[xi].y, drive[xi].x));
  const uint16_t duty = (uint16_t)to_duty(f_amp);
  d_data[xi] = (duty << 8) | phase;
}

void cu_set_from_complex_drive(const hipDoubleComplex* drive, uint32_t size, bool normalize, double max_coefficient, uint16_t* d_data) {
  dim3 block(BLOCK_SIZE, 1, 1);
  dim3 grid((size - 1) / BLOCK_SIZE + 1, 1, 1);
  set_from_complex_drive_kernel<<<grid, block>>>(drive, size, normalize, max_coefficient, d_data);
}

__global__ void set_from_arg(const double* drive, uint32_t size, uint16_t* d_data) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  if (xi >= size) return;

  const uint16_t phase = (uint16_t)to_phase(drive[xi]);
  const uint16_t duty = 0xFF00;
  d_data[xi] = duty | phase;
}

void cu_set_from_arg(const double* drive, uint32_t size, uint16_t* d_data) {
  dim3 block(BLOCK_SIZE, 1, 1);
  dim3 grid((size - 1) / BLOCK_SIZE + 1, 1, 1);
  set_from_arg<<<grid, block>>>(drive, size, d_data);
}

__device__ double3 sub(const double3& a, const double3& b) {
  double x = a.x - b.x;
  double y = a.y - b.y;
  double z = a.z - b.z;
  return double3{x, y, z};
}

__device__ double dot(const double3& a, const double3& b) {
  double x2 = a.x * b.x;
  double y2 = a.y * b.y;
  double z2 = a.z * b.z;
  return x2 + y2 + z2;
}

__device__ double norm(const double3& a) { return sqrt(dot(a, a)); }

__constant__ double DIR_COEF_A[9] = {1.0, 1.0, 1.0, 0.891250938, 0.707945784, 0.501187234, 0.354813389, 0.251188643, 0.199526231};
__constant__ double DIR_COEF_B[9] = {
    0., 0., -0.00459648054721, -0.0155520765675, -0.0208114779827, -0.0182211227016, -0.0122437497109, -0.00780345575475, -0.00312857467007};
__constant__ double DIR_COEF_C[9]{
    0., 0., -0.000787968093807, -0.000307591508224, -0.000218348633296, 0.00047738416141, 0.000120353137658, 0.000323676257958, 0.000143850511};
__constant__ double DIR_COEF_D[9]{
    0., 0., 1.60125528528e-05, 2.9747624976e-06, 2.31910931569e-05, -1.1901034125e-05, 6.77743734332e-06, -5.99548024824e-06, -4.79372835035e-06};

__device__ double t4010a1(double theta_deg) {
  theta_deg = std::abs(theta_deg);
  theta_deg = theta_deg > 90.0 ? 180.0 - theta_deg : theta_deg;
  const auto i = (uint32_t)(ceil(theta_deg / 10.0));
  if (i == 0) return 1.0;
  const auto a = DIR_COEF_A[i - 1];
  const auto b = DIR_COEF_B[i - 1];
  const auto c = DIR_COEF_C[i - 1];
  const auto d = DIR_COEF_D[i - 1];
  const auto x = theta_deg - (double)(i - 1) * 10.0;
  return a + (b + (c + d * x) * x) * x;
}

__device__ hipDoubleComplex transfer(double3& pos, double3& dir, double3 focus, double wavenum, double attenuation) {
  const auto diff = sub(focus, pos);
  const auto dist = norm(diff);
  const auto theta = atan2(dot(diff, dir), dist * norm(dir)) * 180.0 / M_PI;
  const auto directivity = t4010a1(theta);

  const auto v = make_hipDoubleComplex(-dist * attenuation, -wavenum * dist);
  auto r = expc(v);
  r.x *= directivity / dist;
  r.y *= directivity / dist;
  return r;
}

__global__ void transfer_matrix_kernel(const double3* foci, uint32_t foci_num, const double3* positions, const double3* directions,
                                       uint32_t trans_num, double wavenum, double attenuation, hipDoubleComplex* result) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= foci_num || yi >= trans_num) return;

  int dev_idx = yi / core::NUM_TRANS_IN_UNIT;

  double3 focus = foci[xi];
  double3 pos = positions[yi];
  double3 dir = directions[dev_idx];
  result[xi + foci_num * yi] = transfer(pos, dir, focus, wavenum, attenuation);
}

void cu_transfer_matrix(const double3* foci, uint32_t foci_num, const double3* positions, const double3* directions, uint32_t trans_num,
                        double wavenum, double attenuation, hipDoubleComplex* result) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((foci_num - 1) / BLOCK_SIZE + 1, (trans_num - 1) / BLOCK_SIZE + 1, 1);
  transfer_matrix_kernel<<<grid, block>>>(foci, foci_num, positions, directions, trans_num, wavenum, attenuation, result);
}

__global__ void set_bcd_result_kernel(const hipDoubleComplex* vec, uint32_t m, uint32_t idx, hipDoubleComplex* mat) {
  uint32_t xi = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= m || yi >= m) return;

  if (xi == idx) {
    if (yi == idx) return;
    mat[xi + yi * m] = conj(vec[yi]);

  } else if (yi == idx) {
    if (xi == idx) return;
    mat[xi + yi * m] = vec[xi];
  }
}

void cu_set_bcd_result(const hipDoubleComplex* vec, uint32_t m, uint32_t idx, hipDoubleComplex* mat) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((m - 1) / BLOCK_SIZE + 1, (m - 1) / BLOCK_SIZE + 1, 1);
  set_bcd_result_kernel<<<grid, block>>>(vec, m, idx, mat);
}

__global__ void col_sum_abs_kernel(const hipDoubleComplex* din, uint32_t m, uint32_t n, double* dout) {
  extern __shared__ double smem[];

  uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= m) return;

  uint32_t tid = threadIdx.y;
  uint32_t i = blockIdx.y * (blockDim.y * 2) + threadIdx.y;
  double local_sum = (i < n) ? absc(din[i * m + row]) : 0;
  if (i + blockDim.y < n) local_sum += absc(din[(i + blockDim.y) * m + row]);
  smem[tid] = local_sum;
  __syncthreads();

  for (unsigned int s = blockDim.y >> 1; s > 32; s >>= 1) {
    if (tid < s) smem[tid] = local_sum = local_sum + smem[tid + s];
    __syncthreads();
  }
  if (tid < 32) {
    if (blockDim.y >= 64) local_sum += smem[tid + 32];
    for (int offset = 32 >> 1; offset > 0; offset >>= 1) local_sum += __shfl_down_sync(0xffffffff, local_sum, offset);
  }
  if (tid == 0) dout[blockIdx.y * m + row] = local_sum;
}

__global__ void col_sum_kernel(const double* din, uint32_t m, uint32_t n, double* dout) {
  extern __shared__ double smem[];

  uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= m) return;

  uint32_t tid = threadIdx.y;
  uint32_t i = blockIdx.y * (blockDim.y * 2) + threadIdx.y;
  double local_sum = (i < n) ? din[i * m + row] : 0;
  if (i + blockDim.y < n) local_sum += din[(i + blockDim.y) * m + row];
  smem[tid] = local_sum;
  __syncthreads();

  for (unsigned int s = blockDim.y >> 1; s > 32; s >>= 1) {
    if (tid < s) smem[tid] = local_sum = local_sum + smem[tid + s];
    __syncthreads();
  }
  if (tid < 32) {
    if (blockDim.y >= 64) local_sum += smem[tid + 32];
    for (int offset = 32 >> 1; offset > 0; offset >>= 1) local_sum += __shfl_down_sync(0xffffffff, local_sum, offset);
  }
  if (tid == 0) dout[blockIdx.y * m + row] = local_sum;
}

void cu_col_sum_abs(const hipDoubleComplex* transfer, uint32_t m, uint32_t n, double* denominator, double* buffer) {
  dim3 block(1, BLOCK_SIZE / 2, 1);
  dim3 grid(m, (n - 1) / BLOCK_SIZE + 1, 1);

  col_sum_abs_kernel<<<grid, block, BLOCK_SIZE / 2 * sizeof(double)>>>(transfer, m, n, buffer);
  col_sum_kernel<<<dim3(m, 1, 1), dim3(1, grid.y / 2, 1), grid.y / 2 * sizeof(double)>>>(buffer, m, grid.y, denominator);
}

__global__ void make_back_prop_kernel(const hipDoubleComplex* amps, const double* denominator, const hipDoubleComplex* transfer, uint32_t m, uint32_t n,
                                      hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= m || yi >= n) return;

  hipDoubleComplex c = make_hipDoubleComplex(amps[xi].x / denominator[xi], amps[xi].y / denominator[xi]);

  b[yi + n * xi] = mulc(c, conj(transfer[xi + m * yi]));
}

void cu_make_back_prop(const hipDoubleComplex* amps, const double* denominator, const hipDoubleComplex* transfer, uint32_t m, uint32_t n,
                       hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((m - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
  make_back_prop_kernel<<<grid, block>>>(amps, denominator, transfer, m, n, b);
}

__global__ void row_sum_abs_kernel(const hipDoubleComplex* din, const hipDoubleComplex* din2, uint32_t m, uint32_t n, double* dout) {
  extern __shared__ double smem[];

  uint32_t col = blockIdx.y * blockDim.y + threadIdx.y;
  if (col >= n) return;

  uint32_t tid = threadIdx.x;
  uint32_t i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  double local_sum = (i < m) ? absc(mulc(din[i + col * m], din2[i])) : 0;
  if (i + blockDim.x < m) local_sum += absc(mulc(din[i + blockDim.x + col * m], din2[i]));
  smem[tid] = local_sum;
  __syncthreads();

  for (unsigned int s = blockDim.x >> 1; s > 32; s >>= 1) {
    if (tid < s) smem[tid] = local_sum = local_sum + smem[tid + s];
    __syncthreads();
  }
  if (tid < 32) {
    if (blockDim.x >= 64) local_sum += smem[tid + 32];
    for (int offset = 32 >> 1; offset > 0; offset >>= 1) local_sum += __shfl_down_sync(0xffffffff, local_sum, offset);
  }
  if (tid == 0) dout[blockIdx.x + col * m] = local_sum;
}

__global__ void row_sum_kernel(const double* din, uint32_t m, uint32_t n, double* dout) {
  extern __shared__ double smem[];

  uint32_t col = blockIdx.y * blockDim.y + threadIdx.y;
  if (col >= n) return;

  uint32_t tid = threadIdx.x;
  uint32_t i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  double local_sum = (i < m) ? din[i + col * m] : 0;
  if (i + blockDim.x < n) local_sum += din[i + blockDim.x + col * m];
  smem[tid] = local_sum;
  __syncthreads();

  for (unsigned int s = blockDim.x >> 1; s > 32; s >>= 1) {
    if (tid < s) smem[tid] = local_sum = local_sum + smem[tid + s];
    __syncthreads();
  }
  if (tid < 32) {
    if (blockDim.x >= 64) local_sum += smem[tid + 32];
    for (int offset = 32 >> 1; offset > 0; offset >>= 1) local_sum += __shfl_down_sync(0xffffffff, local_sum, offset);
  }
  if (tid == 0) dout[blockIdx.x + col * m] = local_sum;
}

__global__ void transfer_sigma_kernel(double* buffer, uint32_t m, uint32_t n, double gamma, hipDoubleComplex* result) {
  uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;
  if (col >= n) return;

  result[col] = make_hipDoubleComplex(pow(sqrt(buffer[col * m] / m), gamma), 0.0);
}

void cu_make_sigma_diagonal(const hipDoubleComplex* transfer, uint32_t m, uint32_t n, const hipDoubleComplex* amps, double gamma,
                            hipDoubleComplex* result, double* buffer) {
  dim3 block(BLOCK_SIZE / 2, 1, 1);
  dim3 grid((m - 1) / BLOCK_SIZE + 1, n, 1);

  row_sum_abs_kernel<<<grid, block, BLOCK_SIZE / 2 * sizeof(double)>>>(transfer, amps, m, n, buffer);
  row_sum_kernel<<<dim3(1, n, 1), dim3(max(grid.x / 2, 1), 1, 1), max(grid.x / 2, 1) * sizeof(double)>>>(buffer, grid.x, n, buffer);

  transfer_sigma_kernel<<<dim3((n - 1) / BLOCK_SIZE + 1, 1, 1), dim3(BLOCK_SIZE, 1, 1)>>>(buffer, m, n, gamma, result);
}

__global__ void col_sum_imag_kernel(const hipDoubleComplex* din, uint32_t m, uint32_t n, double* dout) {
  extern __shared__ double smem[];

  uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= m) return;

  uint32_t tid = threadIdx.y;
  uint32_t i = blockIdx.y * (blockDim.y * 2) + threadIdx.y;
  double local_sum = (i < n) ? din[i * m + row].y : 0;
  if (i + blockDim.y < n) local_sum += din[(i + blockDim.y) * m + row].y;
  smem[tid] = local_sum;
  __syncthreads();

  for (unsigned int s = blockDim.y >> 1; s > 32; s >>= 1) {
    if (tid < s) smem[tid] = local_sum = local_sum + smem[tid + s];
    __syncthreads();
  }
  if (tid < 32) {
    if (blockDim.y >= 64) local_sum += smem[tid + 32];
    for (int offset = 32 >> 1; offset > 0; offset >>= 1) local_sum += __shfl_down_sync(0xffffffff, local_sum, offset);
  }
  if (tid == 0) dout[blockIdx.y * m + row] = local_sum;
}

void cu_col_sum_imag(const hipDoubleComplex* mat, uint32_t m, uint32_t n, double* result, double* buffer) {
  dim3 block(1, BLOCK_SIZE / 2, 1);
  dim3 grid(m, (n - 1) / BLOCK_SIZE + 1, 1);

  col_sum_imag_kernel<<<grid, block, BLOCK_SIZE / 2 * sizeof(double)>>>(mat, m, n, buffer);
  col_sum_kernel<<<dim3(m, 1, 1), dim3(1, max(grid.y / 2, 1), 1), max(grid.y / 2, 1) * sizeof(double)>>>(buffer, m, grid.y, result);
}
}  // namespace holo
}  // namespace gain
}  // namespace autd
